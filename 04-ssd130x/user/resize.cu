#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

extern "C" {

__global__ void resize_frame_kernel(unsigned char *oframe, int ow, int oh,
				    unsigned char *nframe, int nw, int nh,
				    int threshold, unsigned int *locks)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nw;
	     i += blockDim.x * gridDim.x) {
		for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < nh;
		     j += blockDim.y * gridDim.y) {
			int oi = i * ow / nw;
			int oj = j * oh / nh;

			unsigned char b = oframe[oj * ow * 3 + oi * 3];
			unsigned char g = oframe[oj * ow * 3 + oi * 3 + 1];
			unsigned char r = oframe[oj * ow * 3 + oi * 3 + 2];

			unsigned char brightness =
				r * 0.3 + g * 0.59 + b * 0.11;
			brightness = brightness >= threshold ? 1 : 0;
			brightness = brightness << (j % 8);

			bool leaveloop = false;
			while (!leaveloop) {
				if (atomicExch(&locks[j / 8 * nw + i], 1u) ==
				    0u) {
					nframe[j / 8 * nw + i] |= brightness;
					leaveloop = true;
					atomicExch(&locks[j / 8 * nw + i], 0u);
				}
			}
		}
	}
}

void resize_frame(unsigned char *oframe, int ow, int oh, unsigned char *nframe,
		  int nw, int nh, int threshold)
{
	unsigned char *oframe_d;
	unsigned char *nframe_d;
	unsigned int *locks;

	hipMalloc(&oframe_d, ow * oh * 3);
	hipMalloc(&nframe_d, nw * nh / 8);
	hipMalloc(&locks, sizeof(unsigned int) * nw * nh / 8);

	hipMemcpy(oframe_d, oframe, ow * oh * 3, hipMemcpyHostToDevice);
	hipMemset(nframe_d, 0, nw * nh / 8);
	hipMemset(locks, 0, sizeof(unsigned int) * nw * nh / 8);

	resize_frame_kernel<<<dim3(64, 64), dim3(16, 16)>>>(
		oframe_d, ow, oh, nframe_d, nw, nh, threshold, locks);

	hipMemcpy(nframe, nframe_d, nw * nh / 8, hipMemcpyDeviceToHost);

	hipFree(oframe_d);
	hipFree(nframe_d);
	hipFree(locks);
}
}